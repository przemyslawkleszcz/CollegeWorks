#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

using namespace std;
using namespace cv;

#define BLOCK_SIZE 16

int checkArgumentsNumber(int argc)
{
	int maxArgumentCount = 3;
	if (argc != maxArgumentCount)
	{
		std::cout << "Program wymaga dwóch argumentow.\n";
		return 0;
	}

	return 1;
}

Mat getPicture(char** argv)
{
	Mat picture = imread(argv[1], CV_LOAD_IMAGE_COLOR);
	return picture;
}

int isImageExist(Mat image)
{
	if (!image.data)
	{
		std::cout << "Obraz nie istnieje.\n";
		return 0;
	}

	return 1;
}

__global__
void gaussian(uchar * picture, uchar * pictureNew, long sizeX, long sizeY)
{
	int mask[5][5] = 
	{
		{ 1,4 ,7 ,4 ,1 },
		{ 4,16,26,16,4 },
		{ 7,26,41,26,7 },
		{ 4,16,26,16,4 },
		{ 1,4 ,7 ,4 ,1 }
	};

	int weight = 273;
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < sizeX - 2 && y < sizeY - 2 && x>1 && y>1)
	{
		long r = 0, g = 0, b = 0;
		long wInput, wOutput;

		for (int i_y = 0; i_y < 5; i_y++) 
		{
			for (int i_x = 0; i_x < 5; i_x++) 
			{
				wInput = sizeX * (y + i_y - 2) * 3 + (x + i_x - 2) * 3;
				r += picture[wInput + 2] * mask[i_x][i_y];
				g += picture[wInput + 1] * mask[i_x][i_y];
				b += picture[wInput] * mask[i_x][i_y];
			}
		}

		wOutput = (sizeX - 4)*(y - 2) * 3 + (x - 2) * 3;
		pictureNew[wOutput + 2] = r / weight;
		pictureNew[wOutput + 1] = g / weight;
		pictureNew[wOutput] = b / weight;
	}
}


dim3 getGridDim(Mat picture)
{
	int gridX, gridY;
	gridX = picture.cols / BLOCK_SIZE + 1;
	gridY = picture.rows / BLOCK_SIZE + 1;
	dim3 gridy(gridX, gridY);
	return gridy;
}

int main(int argc, char** argv)
{
	if (!checkArgumentsNumber(argc))
		return -1;

	Mat picture = getPicture(argv);
	if (!isImageExist(picture))
		return -1;

	Mat pictureNew = Mat(picture.rows, picture.cols, CV_8UC3);
	copyMakeBorder(picture, picture, 2, 2, 2, 2, BORDER_REPLICATE);

	dim3 blocks(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grids = getGridDim(picture);

	long sizeIn = sizeof(uchar) * picture.rows* picture.cols * 3;
	long sizeOut = sizeof(uchar) * pictureNew.rows* pictureNew.cols * 3;

	uchar * devPicture;
	uchar * devPictureNew;

	hipMalloc((void**)& devPicture, sizeIn);
	hipMalloc((void**)& devPictureNew, sizeOut);

	if (hipMemcpy(devPicture, picture.data, sizeIn, hipMemcpyHostToDevice) != hipSuccess) 
		cout << "Wystapil blad podczas kopiowania CPU -> GPU\n";

	hipEvent_t timeStart, timeEnd;
	float time;
	hipEventCreate(&timeStart);
	hipEventCreate(&timeEnd);
	hipEventRecord(timeStart, 0);

	gaussian<<<grids, blocks>>>(devPicture, devPictureNew, picture.cols, picture.rows);
	hipDeviceSynchronize();

	hipEventRecord(timeEnd, 0);
	hipEventSynchronize(timeEnd);
	hipEventElapsedTime(&time, timeStart, timeEnd);

	if (hipMemcpy(pictureNew.data, devPictureNew, sizeOut, hipMemcpyDeviceToHost) != hipSuccess) 
		cout << "Wystapil blad podczas kopiowania GPU -> CPU\n";
	
	imwrite(argv[2], pictureNew);
	cout << "Czas: " << time << "ms\n";

	hipFree(devPicture);
	hipFree(devPictureNew);
	return 0;
}

