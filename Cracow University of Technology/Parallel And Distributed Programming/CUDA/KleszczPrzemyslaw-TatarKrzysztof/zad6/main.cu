﻿#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

#define BLOCK_SIZE 100

int checkArgumentsNumber(int argc)
{
	int maxArgumentCount = 2;
	if (argc != maxArgumentCount)
	{
		std::cout << "Program wymaga jednego argumentu.\n";
		return 0;
	}

	return 1;
}

int isFileOk(ifstream& inFile)
{
	if (inFile.fail())
	{
		cout << "Problem z zaladowaniem pliku.\n";
		return 0;
	}

	return 1;
}

std::vector<std::string> getFileLines(ifstream& inFile)
{
	std::vector<std::string> vecOfStr;
	char str[255];
	while (inFile) {
		inFile.getline(str, 255);
		if (inFile)
			vecOfStr.push_back(str);
	}

	return vecOfStr;
}

__global__ void prime(long long int* numbers, int* results)
{
	unsigned int numberIndex = blockIdx.x;
	unsigned long long int number = numbers[numberIndex];

	long long j;
	unsigned long long int sqrtNumber = rint(sqrt((double)number));

	__shared__ 
	bool flag;
	flag = false;

	for (j = (threadIdx.x * (sqrtNumber / blockDim.x) + 2); j < (threadIdx.x * (sqrtNumber / blockDim.x)) + (sqrtNumber / blockDim.x) + 2; j++)
	{
		__syncthreads();
		if (flag)
			continue;

		if (number % j == 0)
		{
			flag = true;
			results[numberIndex] = 0;
		}
	}

	if (!flag)
		results[numberIndex] = 1;
}


int main(int argc, char *argv[])
{
	if (!checkArgumentsNumber(argc))
		return -1;

	ifstream inFile;
	inFile.open(argv[1]);
	int isOk = isFileOk(inFile);
	if (!isOk)
		return -1;

	std::vector<std::string> vecOfStr = getFileLines(inFile);
	std::vector<std::string> vecOfRes;

	long long int* tabOfNumbers_l = new long long int[vecOfStr.size()];
	for (int i = 0; i < vecOfStr.size(); i++)
		tabOfNumbers_l[i] = atoll(vecOfStr[i].c_str());

	long long int* devNumbers;
	hipMalloc((void**)&devNumbers, vecOfStr.size() * sizeof(long long int));
	int* devResults;
	hipMalloc((void**)&devResults, vecOfStr.size() * sizeof(int));

	hipMemcpy(devNumbers, tabOfNumbers_l, vecOfStr.size() * sizeof(long long int), hipMemcpyHostToDevice);
	int* results = new int[vecOfStr.size()];
	hipMemset(devResults, 0, vecOfStr.size() * sizeof(int));

	hipEvent_t timeStart, timeEnd;
	float time;
	hipEventCreate(&timeStart);
	hipEventCreate(&timeEnd);
	hipEventRecord(timeStart, 0);

	prime<<<vecOfStr.size(),BLOCK_SIZE>>>(devNumbers, devResults);

	hipDeviceSynchronize();
	hipEventRecord(timeEnd, 0);
	hipEventSynchronize(timeEnd);
	hipEventElapsedTime(&time, timeStart, timeEnd);

	cout << "Time: " << time << "ms\n";
	hipMemcpy(results, devResults, vecOfStr.size() * sizeof(int), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < vecOfStr.size(); i++)
		cout << vecOfStr[i] << ": " << (results[i] ? "prime" : "composite") << "\n";

	hipFree(devNumbers);
	hipFree(devResults);
	free(tabOfNumbers_l);
	free(results);

	return 0;
}