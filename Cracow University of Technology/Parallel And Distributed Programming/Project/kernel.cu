﻿#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <fstream>
#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hiprand/hiprand.h>

using namespace std;

#define N 9
#define n 3
#define STEPS 18

int *devNewSudokus;
int *devOldSudokus;
int *devGaps;
int *devGapsNumber;
int *devBoardIndex;

int *devIsCompleted;
int *devResult;

int *board = new int[N * N];
int *result = new int[N * N];

int checkArgumentsNumber(int argc)
{
	int maxArgumentCount = 4;
	if (argc != maxArgumentCount)
	{
		std::cout << "Program wymaga trzech argumentów.\n";
		return 0;
	}

	return 1;
}

int loadSudoku(char *nameOfFile, int *board)
{
	FILE * fileToRead = fopen(nameOfFile, "r");
	if (fileToRead == NULL)
	{
		printf("Problem z zaladowaniem pliku wejsciowego.\n");
		return 0;
	}

	char number;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (!fscanf(fileToRead, "%c\n", &number))
			{
				printf("Blad podczas ladowania pliku wejsciowego\n");
				return 0;
			}

			if (number >= '1' && number <= '9')
				board[i * N + j] = (int)(number - '0');
			else
				board[i * N + j] = 0;
		}
	}

	return 1;
}

void initializeCudaMemoryBfs()
{
	const int sk = pow(2, 26);
	hipMalloc(&devGaps, sk * sizeof(int));
	hipMalloc(&devGapsNumber, (sk / 81 + 1) * sizeof(int));
	hipMalloc(&devNewSudokus, sk * sizeof(int));
	hipMalloc(&devOldSudokus, sk * sizeof(int));
	hipMalloc(&devBoardIndex, sizeof(int));

	hipMemset(devBoardIndex, 0, sizeof(int));
	hipMemset(devNewSudokus, 0, sk * sizeof(int));
	hipMemset(devOldSudokus, 0, sk * sizeof(int));
}

void initializeCudaMemoryBacktrack() 
{
	hipMalloc(&devIsCompleted, sizeof(int));
	hipMalloc(&devResult, N * N * sizeof(int));
	hipMemset(devIsCompleted, 0, sizeof(int));
	hipMemcpy(devResult, board, N * N * sizeof(int), hipMemcpyHostToDevice);
}

void freeMemory()
{
	hipFree(devGaps);
	hipFree(devGapsNumber);
	hipFree(devNewSudokus);
	hipFree(devOldSudokus);
	hipFree(devBoardIndex);
	hipFree(devIsCompleted);
	hipFree(devResult);

	delete[] board;
	delete[] result;
}

void printBoard(int *board)
{
	for (int i = 0; i < N; i++)
	{
		if (i % n == 0)
			printf("-----------------------\n");

		for (int j = 0; j < N; j++)
		{
			if (j % n == 0)
				printf("| ");

			printf("%d ", board[i * N + j]);
		}

		printf("|\n");
	}

	printf("-----------------------\n");
}

__device__
void resetBitmap(bool *bitMap, int length) 
{
	for (int i = 0; i < length; i++) 
		bitMap[i] = false;
}

__device__
bool checkRow(const int *board, bool* seen, int row)
{
	for (int i = 0; i < N; i++) 
	{
		int val = board[row * N + i];
		if (val != 0)
		{
			if (seen[val - 1])
				return false;
			else
				seen[val - 1] = true;
		}
	}

	return true;
}

__device__
bool checkColumn(const int *board, bool* seen, int col)
{
	for (int j = 0; j < N; j++)
	{
		int val = board[j * N + col];
		if (val != 0)
		{
			if (seen[val - 1])
				return false;
			else
				seen[val - 1] = true;
		}
	}

	return true;
}

__device__
bool checkSubBoard(const int *board, bool* seen, int row, int col)
{
	int ridx = row / n;
	int cidx = col / n;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			int val = board[(ridx * n + i) * N + (cidx * n + j)];
			if (val != 0)
			{
				if (seen[val - 1])
					return false;
				else
					seen[val - 1] = true;
			}
		}
	}

	return true;
}

__device__
bool validBoard(const int *board, int index) 
{
	int r = index / 9;
	int c = index % 9;

	if ((board[index] < 1) || (board[index] > 9))
		return false;

	bool seen[N];
	resetBitmap(seen, N);

	if (!checkRow(board, seen, r))
		return false;

	resetBitmap(seen, N);

	if (!checkColumn(board, seen, c))
		return false;

	resetBitmap(seen, N);

	if (!checkSubBoard(board, seen, r, c))
		return false;

	//ok
	return true;
}

__global__
void Backtrack(int* newSudokus,
	const int numberOfPossibleBoards,
	int* gaps,
	int* gapsNumber,
	int* isCompleted,
	int* result) 
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int *currentBoard;
	int *currentEmptySpaces;
	int currentNumEmptySpaces;

	while ((*isCompleted == 0) && (tid < numberOfPossibleBoards))
	{
		int emptyIndex = 0;
		currentBoard = newSudokus + tid * 81;
		currentEmptySpaces = gaps + tid * 81;
		currentNumEmptySpaces = gapsNumber[tid];

		while ((emptyIndex >= 0) && (emptyIndex < currentNumEmptySpaces)) 
		{
			//wyciągamy indeks miejsca 0 na podstawie kolekcji currentEmptySpaces
			//wstawiamy następna liczbę w puste miejsce
			currentBoard[currentEmptySpaces[emptyIndex]]++;

			if (!validBoard(currentBoard, currentEmptySpaces[emptyIndex])) 
			{
				if (currentBoard[currentEmptySpaces[emptyIndex]] >= 9) 
				{
					//jesli wychodzi poza zakres pól - backtrack
					currentBoard[currentEmptySpaces[emptyIndex]] = 0;
					emptyIndex--;
				}
			}
			else
				emptyIndex++;
		}

		if (emptyIndex == currentNumEmptySpaces) 
		{
			*isCompleted = 1; // znaleziono
			for (int i = 0; i < N * N; i++)
				result[i] = currentBoard[i];
		}

		tid += gridDim.x * blockDim.x;
	}
}

__device__
void testColumns(int* previousSudokus, int row, int attempt, unsigned int index, int* works) {
	for (int column = 0; column < N; column++)
		if (previousSudokus[row * N + column + N * N * index] == attempt)
			*works = 0;
}

__device__
void testRows(int* previousSudokus, int column, int attempt, unsigned int index, int* works) {
	for (int row = 0; row < N; row++)
		if (previousSudokus[(row * N) + column + (N * N * index)] == attempt)
			*works = 0;
}

__device__
void testBlock(int* previousSudokus, int col, int row, int attempt, unsigned int index, int* works)
{
	for (int r = n * (row / n); r < n; r++)
	{
		for (int c = n * (col / n); c < n; c++)
			if (previousSudokus[r * N + c + N * N * index] == attempt)
				*works = 0;
	}
}

__device__
void copySudoku(
	int* previousSudokus,
	int* newSudokus,
	int* boardIndex,
	int* gaps,
	int* gapsNumber,
	int row,
	int col,
	int tid,
	int attempt)
{
	int nextBoardIndex = atomicAdd(boardIndex, 1);
	int emptyIndex = 0;
	for (int r = 0; r < 9; r++)
	{
		for (int c = 0; c < 9; c++)
		{
			newSudokus[nextBoardIndex * 81 + r * 9 + c] = previousSudokus[tid * 81 + r * 9 + c];
			if (previousSudokus[tid * 81 + r * 9 + c] == 0 && (r != row || c != col))
			{
				gaps[emptyIndex + 81 * nextBoardIndex] = r * 9 + c;
				emptyIndex++;
			}
		}
	}

	gapsNumber[nextBoardIndex] = emptyIndex;
	newSudokus[nextBoardIndex * 81 + row * 9 + col] = attempt;
}

__global__
void
Bfs(int* oldSudokus,
	int* newSudokus,
	int totalSudokus,
	int* boardIndex,
	int* gaps,
	int* gapsNumber)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	//dywergencja
	while (tid < totalSudokus)
	{
		int found = 0;
		for (int i = (tid * N * N); (i < (tid * N * N) + N * N) && (found == 0); i++)
		{
			if (oldSudokus[i] == 0) //znaleziono
			{
				found = 1;
				int temp = i - N * N * tid;
				int row = temp / N;
				int col = temp % N;

				for (int attempt = 1; attempt <= N; attempt++)
				{
					int works = 1;
					testColumns(oldSudokus, row, attempt, tid, &works);
					testRows(oldSudokus, col, attempt, tid, &works);
					testBlock(oldSudokus, col, row, attempt, tid, &works);

					if (works == 1)
						copySudoku(oldSudokus, newSudokus, boardIndex, gaps, gapsNumber, row, col, tid, attempt);
				}
			}
		}

		tid += blockDim.x * gridDim.x;
	}
}

int findPossibleBoards(unsigned int blocks, unsigned int threads)
{
	int numberOfPossibleBoards;
	int totalSudokus = 1;

	hipMemcpy(devOldSudokus, board, N * N * sizeof(int), hipMemcpyHostToDevice);
	Bfs<<<blocks, threads>>>(devOldSudokus, devNewSudokus, totalSudokus, devBoardIndex, devGaps, devGapsNumber);

	for (int i = 0; i < STEPS; i++)
	{
		hipMemcpy(&numberOfPossibleBoards, devBoardIndex, sizeof(int), hipMemcpyDeviceToHost);
		hipMemset(devBoardIndex, 0, sizeof(int));

		if (i % 2 == 0)
			Bfs<<<blocks, threads>>>(devNewSudokus, devOldSudokus, numberOfPossibleBoards, devBoardIndex, devGaps, devGapsNumber);
		else
			Bfs<<<blocks, threads>>>(devOldSudokus, devNewSudokus, numberOfPossibleBoards, devBoardIndex, devGaps, devGapsNumber);
	}

	hipMemcpy(&numberOfPossibleBoards, devBoardIndex, sizeof(int), hipMemcpyDeviceToHost);
	printf("Liczba uzyskanych wariantów konfiguracji: %d\n", numberOfPossibleBoards);
	return numberOfPossibleBoards;
}

int main(int argc, char* argv[])
{
	if (!checkArgumentsNumber(argc))
		return -1;

	int blocks = atoi(argv[1]);
	int threads = atoi(argv[2]);
	char* nameOfFile = argv[3];

	if (!loadSudoku(nameOfFile, board))
		return -1;

	initializeCudaMemoryBfs();
	initializeCudaMemoryBacktrack();

	hipEvent_t timeStart, timeEnd;
	float time;
	hipEventCreate(&timeStart);
	hipEventCreate(&timeEnd);
	hipEventRecord(timeStart, 0);

	int numberOfPossibleBoards = findPossibleBoards(blocks, threads);

	if (STEPS % 2 == 1)
		devNewSudokus = devOldSudokus;

	Backtrack<<<blocks, threads>>>(devNewSudokus, numberOfPossibleBoards, devGaps, devGapsNumber, devIsCompleted, devResult);

	hipDeviceSynchronize();
	hipEventRecord(timeEnd, 0);
	hipEventSynchronize(timeEnd);
	hipEventElapsedTime(&time, timeStart, timeEnd);

	memset(result, 0, N * N * sizeof(int));
	hipMemcpy(result, devResult, N * N * sizeof(int), hipMemcpyDeviceToHost);
	printBoard(result);
	cout << "Time: " << time << "ms\n";

	freeMemory();
	return 0;
}
